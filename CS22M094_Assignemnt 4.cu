	#include <iostream>
	#include <stdio.h>
	#include <hip/hip_runtime.h>

	#define max_N 100000
	#define max_P 30
	#define BLOCKSIZE 1024

	using namespace std;




//****************Kernels start here***************************

//kernel for val_initial the lock array
__global__ void val_initial(int target,int *lock)
	{
    unsigned id =threadIdx.x-1+blockIdx.x*BLOCKSIZE;
		if(id+1<target) lock[id+1]=10000;
	}


	//kernel to calculate the request processed by computer center
	__global__ void assignwork(int start,float rn,int R,int *d_var_lock,float z,int *d_centre,int *d_facility,int *d_offset,int *d_totalcapacity,int volatile *d_totalslots,int *d_neces_cen,int * d_facility_offset,int *d_neces_start,int *d_neces_fac,unsigned int *d_total_succ_reqs,int *d_neces_slots)
	{
 
     	__shared__ unsigned total;
		   int n;
       int p=0;
		   int id=start+threadIdx.x+blockIdx.x*BLOCKSIZE;
		
	

    int total_house;            //variable holding the total no of centers
    total_house=d_neces_cen[id];
    total=1;
    int slot_no;
    slot_no=d_neces_fac[id]*24+d_offset[total_house];
    int choose;
    choose=d_neces_fac[id]+d_facility_offset[total_house];
    int temp=0;
    int perform=1;
  __syncthreads();            //to synchronise all the requests

	
    int c=-1+d_neces_start[id];
    int d=d_neces_slots[id]-2+d_neces_start[id];

			
   
	    do{
      total=1;
	    if(perform!=p) atomicMin(&(d_var_lock[choose]),id);          //to find out the minimum request id and serve it first
      

		__syncthreads();
			
	    if(perform!=0)
			{			
         if(id==d_var_lock[choose])
				{
				for(int i=c+slot_no;i<=(d+slot_no);i++)
					{
				      d_totalslots[i]=-1+d_totalslots[i];
              if(d_totalslots[i]<p)
						{	
							n = i;
							temp = 1;
							break;						
						}
					}
					
					if(temp!=0)
					{

			         for(int i = (c+slot_no); i <= n; i++)
						{
							d_totalslots[i]=1+d_totalslots[i];
						}
						temp = p;
					}
					else
					{	   int val=1+total_house;
				        atomicInc(&d_total_succ_reqs[val],R);
			            atomicInc(&d_total_succ_reqs[0],R);
			        }
					
					
					perform=p;
					d_var_lock[choose] = 1000000;	
				}

				else if(perform!=0) total=p;
					
			}
			__syncthreads();
					
		}while(total==p);
		__syncthreads();
		
	}

//*********************Kernels end here**************************


int main(int argc,char **argv)
{
// variable declarations...
    int N,*total_house,*facility,*capacity,*fac_ids, *succ_reqs, *tot_reqs;
    

    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");

    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0; 
    }

    fscanf( inputfilepointer, "%d", &N ); // N is number of centres
	
    // Allocate memory on cpu
    total_house=(int*)malloc(N * sizeof (int));  // Computer  total_house numbers
    facility=(int*)malloc(N * sizeof (int));  // Number of facilities in each computer total_house
    fac_ids=(int*)malloc(max_P * N  * sizeof (int));  // Facility room numbers of each computer total_house
    capacity=(int*)malloc(max_P * N * sizeof (int));  // stores capacities of each facility for every computer total_house 


    int success=0;  // total successful requests
    int fail = 0;   // total failed requests
    tot_reqs = (int *)malloc(N*sizeof(int));   // total requests for each total_house
    succ_reqs = (int *)malloc((N+1)*sizeof(int)); // total successful requests for each total_house

    // Input the computer centres data
    int k1=0 , k2 = 0;
    for(int i=0;i<N;i++)
    {
      fscanf( inputfilepointer, "%d", &total_house[i] );
      fscanf( inputfilepointer, "%d", &facility[i] );
      
      for(int j=0;j<facility[i];j++)
      {
        fscanf( inputfilepointer, "%d", &fac_ids[k1] );
        k1++;
      }
      for(int j=0;j<facility[i];j++)
      {
        fscanf( inputfilepointer, "%d", &capacity[k2]);
        k2++;     
      }
    }

    // variable declarations
    int *req_id, *req_cen, *req_fac, *req_start, *req_slots;   // Number of total_slots requested for every request
    
    // Allocate memory on CPU 
	  int R;
	  fscanf( inputfilepointer, "%d", &R); // Total requests
    req_id = (int *) malloc ( (R) * sizeof (int) );  // Request ids
    req_cen = (int *) malloc ( (R) * sizeof (int) );  // Requested computer total_house
    req_fac = (int *) malloc ( (R) * sizeof (int) );  // Requested facility
    req_start = (int *) malloc ( (R) * sizeof (int) );  // Start slot of every request
    req_slots = (int *) malloc ( (R) * sizeof (int) );   // Number of total_slots requested for every request
    
    // Input the user request data
    for(int j = 0; j < R; j++)
    {
       fscanf( inputfilepointer, "%d", &req_id[j]);
       fscanf( inputfilepointer, "%d", &req_cen[j]);
       fscanf( inputfilepointer, "%d", &req_fac[j]);
       fscanf( inputfilepointer, "%d", &req_start[j]);
       fscanf( inputfilepointer, "%d", &req_slots[j]);
       tot_reqs[req_cen[j]]+=1;  
    }


	//*************************Kernel calling start**************************************************
 
    int temp1=0;      //temporary variables used
    int temp2=0;
    int *total_slots;
    total_slots=(int*)malloc(max_P*N*(24)*sizeof(int));
	  int *facility_no;
	  facility_no=(int*)malloc(N*sizeof(int));
	  int *off_value;
    off_value=(int*)malloc(N*sizeof(int));
	 
  	unsigned int *d_total_succ_reqs;       //to maintain the total succ request at device
	
     //memory allocation at GPU for all the required variables
    int *d_totalslots;
		hipMalloc(&d_totalslots,(max_P*N*24)*sizeof(int));
		int *d_totalcapacity;
		hipMalloc(&d_totalcapacity,(max_P*N)*sizeof(int));
    int *d_facility_offset;
		hipMalloc(&d_facility_offset,(max_P*N)*sizeof(int));
		int *d_total_off;
		hipMalloc(&d_total_off,(N)*sizeof(int));	
		int *d_centre;
		hipMalloc(&d_centre,(N)*sizeof(int));
		int *d_facility;
		hipMalloc(&d_facility,(N)*sizeof(int));
	
		int *d_var_lock;
	  int i=0;
    while(i<N)
          {
             off_value[i]=temp2;
		         facility_no[i]=temp1;
		  
      for(int j=0;j<facility[i];j++)
		  { int k=temp2;
		    while(k<temp2+24){
			  total_slots[k]=capacity[temp1];
			  k++;
		     }
		  temp2=temp2+24;
		  temp1=temp1+1;}
		i++;
		}
    // Copy memory from host to device for all the required variables
		hipMemcpy(d_totalslots,total_slots,temp2*sizeof(int),hipMemcpyHostToDevice);
	  hipMemcpy(d_total_off,off_value,N*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_facility_offset,facility_no,temp2*sizeof(int),hipMemcpyHostToDevice);
		hipMemcpy(d_facility,facility,N*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_totalcapacity,capacity,temp1*sizeof(int),hipMemcpyHostToDevice);
		hipMemcpy(d_centre,total_house,N*sizeof(int),hipMemcpyHostToDevice);
		
    
		int initialiseblock=ceil((float(temp1)/BLOCKSIZE));
		
		hipMalloc(&d_var_lock,(temp1)*sizeof(int));
    //kernel calling to initialize the lock array
		val_initial<<<initialiseblock,BLOCKSIZE>>>(temp1,d_var_lock);	
    
        int totalblocks=R/BLOCKSIZE;     //calculating total no of blocks required
        int *d_neces_fac;
        hipMalloc(&d_neces_fac,(R)*sizeof(int));
		    hipMemcpy(d_neces_fac,req_fac,R*sizeof(int),hipMemcpyHostToDevice);
			
     //memory allocation to device variables and copying memory
      int *d_neces_cen;
			hipMalloc(&d_neces_cen,(R)*sizeof(int));
			hipMemcpy(d_neces_cen,req_cen,R*sizeof(int),hipMemcpyHostToDevice);
            
			int *d_neces_start;
			hipMalloc(&d_neces_start,(R)*sizeof(int));
			hipMemcpy(d_neces_start,req_start,R*sizeof(int),hipMemcpyHostToDevice);
 
      int *d_neces_slots;
			hipMalloc(&d_neces_slots,(R)*sizeof(int));
			hipMemcpy(d_neces_slots,req_slots,R*sizeof(int),hipMemcpyHostToDevice);

			hipMalloc(&d_total_succ_reqs,(N+1)*sizeof(int));
			hipMemcpy(d_total_succ_reqs,succ_reqs,(N+1)* sizeof(int),hipMemcpyHostToDevice);
			
			
      int count=0;   //maintains the no of kernel calling
			int extrathread=R%BLOCKSIZE;
      
			for(int x=totalblocks;x>0;x--){
              count++;
            assignwork<<<1,BLOCKSIZE>>>(BLOCKSIZE*(totalblocks-x),0,R,d_var_lock,0,d_centre,d_facility,d_total_off,d_totalcapacity,d_totalslots,d_neces_cen,d_facility_offset,d_neces_start,d_neces_fac,d_total_succ_reqs,d_neces_slots);
				   count=count+x;
         hipDeviceSynchronize();
			}
			

       //After counting count,1 more time calling assignwork required to complete the required task
			assignwork<<<1,extrathread>>>(BLOCKSIZE*(totalblocks),0,R,d_var_lock,0,d_centre,d_facility,d_total_off,d_totalcapacity,d_totalslots,d_neces_cen,d_facility_offset,d_neces_start,d_neces_fac,d_total_succ_reqs,d_neces_slots);
      count++;		
			hipMemcpy(succ_reqs,d_total_succ_reqs,(N+1)*sizeof(int),hipMemcpyDeviceToHost);

    
   
   //*****************************Kernel calling end***************************************************



    // Output
    char *outputfilename = argv[2]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");

    fprintf( outputfilepointer, "%d %d\n",succ_reqs[0], R-succ_reqs[0]);
    for(int j = 0; j < N; j++)
    {
        fprintf( outputfilepointer, "%d %d\n", succ_reqs[j+1], tot_reqs[j]-succ_reqs[j+1]);
    }
    fclose( inputfilepointer );
    fclose( outputfilepointer );
    hipDeviceSynchronize();
	  return 0;
			
	}
